#include "hip/hip_runtime.h"
/*
 * Implementing Breadth first search on CUDA using algorithm given in HiPC'07
 * paper "Accelerating Large Graph Algorithms on the GPU using CUDA"
 *
 * Copyright (c) 2008 
 * International Institute of Information Technology - Hyderabad. 
 * All rights reserved.
 *
 * Permission to use, copy, modify and distribute this software and its 
 * documentation for educational purpose is hereby granted without fee, 
 * provided that the above copyright notice and this permission notice 
 * appear in all copies of this software and that you do not sell the software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, 
 * IMPLIED OR OTHERWISE.
 *
 * Created by Pawan Harish.
 *
 * Modified by Shinpei Kato.
 */

#define NUM_ITR 1000000

__global__ void
Kernel2(int *g_graph_mask, int *g_updating_graph_mask, int *g_graph_visited, int *g_over, int no_of_nodes)
{
for (int itr = 0; itr < NUM_ITR; ++itr) {

	int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;
	if (tid < no_of_nodes) {
		if (g_updating_graph_mask[tid]) {
			g_graph_mask[tid] = true;
			g_graph_visited[tid] = true;
			*g_over = true;
			g_updating_graph_mask[tid] = false;
		}
	}

}
}
